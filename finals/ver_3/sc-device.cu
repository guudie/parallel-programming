#include "hip/hip_runtime.h"
#include "device.cuh"

__global__ void computeEnergyKernel(const uchar3* inPixels, int* energy, int width, int height) {
    extern __shared__ uchar3 s_inPixels[];
    int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int dimXWithFilter = blockDim.x + 2;
    uchar3 zero_uchar3 = {0, 0, 0};
	s_inPixels[(threadIdx.y + 1) * dimXWithFilter + threadIdx.x + 1] = (r < height && c < width) ? inPixels[r * width + c] : zero_uchar3;
	if(threadIdx.y < 1) {
		// top apron
		int rt = r - 1;
		s_inPixels[threadIdx.y * dimXWithFilter + threadIdx.x + 1] = (rt >= 0 && c < width) ? inPixels[rt * width + c] : zero_uchar3;

		// bottom apron
		int rb = (blockIdx.y + 1) * blockDim.y + threadIdx.y;
		s_inPixels[(threadIdx.y + blockDim.y + 1) * dimXWithFilter + threadIdx.x + 1] = (rb < height && c < width) ? inPixels[rb * width + c] : zero_uchar3;

		// left & right aprons
		int cl = blockIdx.x * blockDim.x - 1 + threadIdx.y;
		int cr = (blockIdx.x + 1) * blockDim.x + threadIdx.y;
        for(int idx = threadIdx.x; idx < blockDim.y + 2; idx += blockDim.x) {
            int tmpR = blockIdx.y * blockDim.y + idx - 1;
		    s_inPixels[idx * dimXWithFilter + threadIdx.y] = (cl >= 0 && 0 <= tmpR && tmpR < height) ? inPixels[tmpR * width + cl] : zero_uchar3;
		    s_inPixels[idx * dimXWithFilter + blockDim.x + 1 + threadIdx.y] = (cr < width && 0 <= tmpR && tmpR < height) ? inPixels[tmpR * width + cr] : zero_uchar3;
        }
	}
	__syncthreads();

    if(r < height && c < width) {
		int x = 0, y = 0;
		for(int f_r = 0; f_r < 3; f_r++) {
			for(int f_c = 0; f_c < 3; f_c++) {
				int ri = threadIdx.y + f_r;
				int ci = threadIdx.x + f_c;
				uchar3 val = s_inPixels[ri * dimXWithFilter + ci];
				x += (val.x + val.y + val.z) / 3 * d_xSobel[f_r * 3 + f_c];
				y += (val.x + val.y + val.z) / 3 * d_ySobel[f_r * 3 + f_c];
			}
		}
		energy[r * width + c] = abs(x) + abs(y);
	}
}

// called with 1 flat block
__global__ void computeSeamsKernel(const int* energy, int2* dp, int width, int height) {
    extern __shared__ int s_rows[]; // stores 2 consecutive rows for faster memory access
    for(int c = threadIdx.x; c < width; c += blockDim.x) {
        dp[c] = make_int2(energy[c], 0);
        s_rows[width + c] = energy[c];
    }
    __syncthreads();
    for(int r = 1; r < height; r++) {
        for(int c = threadIdx.x; c < width; c += blockDim.x) {
            int i = r * width + c;
            int2 res = make_int2(s_rows[(r & 1) * width + c], c);
            if(c - 1 >= 0)
                if(res.x >= s_rows[(r & 1) * width + c - 1])
                    res = make_int2(s_rows[(r & 1) * width + c - 1], c - 1);
            if(c + 1 < width)
                if(res.x > s_rows[(r & 1) * width + c + 1])
                    res = make_int2(s_rows[(r & 1) * width + c + 1], c + 1);
            res.x += energy[i];
            dp[i] = res;
            s_rows[(1 - (r & 1)) * width + c] = res.x;
        }
        __syncthreads();
    }
}

__global__ void minReductionKernel(const int2* dp_lastRow, int width, int2* blockMin) {
    extern __shared__ int2 s_data[];
    int c = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    if(c < width)
        s_data[threadIdx.x] = make_int2(dp_lastRow[c].x, c);
    if(c + blockDim.x < width)
        s_data[threadIdx.x + blockDim.x] = make_int2(dp_lastRow[c + blockDim.x].x, c + blockDim.x);
    __syncthreads();

    for(int stride = blockDim.x; stride > 0; stride /= 2) {
        if(threadIdx.x < stride) {
            int2& a = s_data[threadIdx.x];
            int2 b = s_data[threadIdx.x + stride];
            if(c + stride < width && (a.x > b.x || (a.x == b.x && a.y > b.y)))
                a = b;
        }
        __syncthreads();
    }

    if(threadIdx.x == 0)
        blockMin[blockIdx.x] = s_data[0];
}

// inPixels1 contains pixel info for the current iteration, inPixels2 will be calculated to hold info for the next
__global__ void carveSeamKernel(uchar3* inPixels1, uchar3* inPixels2, int* trace, int width, int height) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if(r < height && c < width - 1)
        inPixels2[r * (width - 1) + c] = inPixels1[r * width + c + (c >= trace[r])];
}

void seamCarvingGpu(const uchar3* inPixels, uchar3* outPixels, int width, int height, int targetWidth,
        int* xSobel, int* ySobel, dim3 blockSize1D, dim3 blockSize2D)
{
    dim3 blockSizeEnergy = blockSize2D;
    dim3 blockSizeSeams = blockSize1D;
    dim3 blockSizeReduction = blockSize1D;
    dim3 blockSizeCarve = blockSize2D;
    ///////////////////////////////////////////

    uchar3 *d_inPixels1, *d_inPixels2;
    int *d_energy, *d_trace;
    int2 *d_dp, *d_blockMin;

    int *trace;
    int2 *dp, *blockMin;

    size_t inSize = sizeof(uchar3) * width * height;
    size_t arrSize = sizeof(int) * width * height;

    CHECK(hipMalloc(&d_inPixels1, inSize));
    CHECK(hipMalloc(&d_inPixels2, inSize));

    CHECK(hipMalloc(&d_energy, arrSize));
    CHECK(hipMalloc(&d_trace, arrSize));
    CHECK(hipMalloc(&d_dp, sizeof(int2) * width * height));
    CHECK(hipMalloc(&d_blockMin, sizeof(int2) * ((width - 1) / blockSizeReduction.x / 2 + 1)));

    dp = (int2*)malloc(sizeof(int2) * width * height);
    trace = (int*)malloc(sizeof(int) * height);
    blockMin = (int2*)malloc(sizeof(int2) * ((width - 1) / blockSizeReduction.x / 2 + 1));

    CHECK(hipMemcpy(d_inPixels1, inPixels, inSize, hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_xSobel), xSobel, sizeof(int) * 9));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_ySobel), ySobel, sizeof(int) * 9));

    hipStream_t streams[2];
    for(int i = 0; i < 2; i++)
        CHECK(hipStreamCreate(streams + i));
    CHECK(hipHostRegister(dp, sizeof(int2) * width * height, hipHostRegisterDefault));
    CHECK(hipHostRegister(trace, sizeof(int) * height, hipHostRegisterDefault));

    for(int curWidth = width; curWidth > targetWidth; curWidth--) {
        dim3 gridSizeEnergy((curWidth - 1) / blockSizeEnergy.x + 1, (height - 1) / blockSizeEnergy.y + 1);
        dim3 gridSizeSeams(1);
        dim3 gridSizeReduction((curWidth - 1) / blockSizeReduction.x / 2 + 1);
        dim3 gridSizeCarve((curWidth - 1) / blockSizeCarve.x + 1, (height - 1) / blockSizeCarve.y + 1);

        int smemEnergy = (blockSizeEnergy.x + 2) * (blockSizeEnergy.y + 2) * sizeof(uchar3);
        int smemSeams = 2 * curWidth * sizeof(int);
        int smemReduction = 2 * blockSizeReduction.x * sizeof(int2);

        // compute energy
        computeEnergyKernel<<<gridSizeEnergy, blockSizeEnergy, smemEnergy>>>(d_inPixels1, d_energy, curWidth, height);
        // dynamic programming
        computeSeamsKernel<<<gridSizeSeams, blockSizeSeams, smemSeams>>>(d_energy, d_dp, curWidth, height);
        // reduction to find min
        minReductionKernel<<<gridSizeReduction, blockSizeReduction, smemReduction, streams[0]>>>(d_dp + (height - 1) * curWidth, curWidth, d_blockMin);
        CHECK(hipMemcpyAsync(blockMin, d_blockMin, sizeof(int2) * gridSizeReduction.x, hipMemcpyDeviceToHost, streams[0]));

        CHECK(hipMemcpyAsync(dp, d_dp, sizeof(int2) * curWidth * height, hipMemcpyDeviceToHost, streams[1]));

        hipDeviceSynchronize();

        int2 res = blockMin[0];
        for(int i = 1; i < gridSizeReduction.x; i++)
            if(res.x > blockMin[i].x)
                res = blockMin[i];
        trace[height - 1] = res.y;

        // tracing
        for(int r = height - 1; r > 0; r--) {
            trace[r - 1] = dp[r * curWidth + trace[r]].y;
        }

        CHECK(hipMemcpy(d_trace, trace, sizeof(int) * height, hipMemcpyHostToDevice));
        // remove seam
        carveSeamKernel<<<gridSizeCarve, blockSizeCarve>>>(d_inPixels1, d_inPixels2, d_trace, curWidth, height);

        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        swapPtr(d_inPixels1, d_inPixels2);
    }

    CHECK(hipMemcpy(outPixels, d_inPixels1, sizeof(uchar3) * targetWidth * height, hipMemcpyDeviceToHost));

    for(int i = 0; i < 2; i++)
        CHECK(hipStreamDestroy(streams[i]));

    CHECK(hipFree(d_inPixels1));
    CHECK(hipFree(d_inPixels2));
    CHECK(hipFree(d_energy));
    CHECK(hipFree(d_trace));
    CHECK(hipFree(d_dp));
    CHECK(hipFree(d_blockMin));
    free(trace);
    free(dp);
    free(blockMin);
}
